#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>

hipError_t softmaxWithCuda(float* input, float* output, int width, int height);

__global__ void softmaxkernel(float* input, float* output, int width, int height) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    int index = row * width + col;

    if (row < height && col < width) {
        __shared__ float shared_value[1024];
        __shared__ float shared_sum;

        float exp_value = expf(input[index]);
        shared_value[col] = exp_value;

        __syncthreads();

        if (col == 0) {
            float sum = 0.0f;
            for (int i = 0; i < width; ++i) {
                sum += shared_value[i];
            }
            shared_sum = sum;
        }

        __syncthreads();

        output[index] = shared_value[col] / shared_sum;
    }
}

hipError_t softmaxWithCuda(float* input, float* output, int width, int height) {
    float* dev_input = nullptr;
    float* dev_output = nullptr;
    hipError_t cudaStatus;

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_input, width * height * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_input failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_output, width * height * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_output failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy input data from host to device
    cudaStatus = hipMemcpy(dev_input, input, width * height * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy input failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Launch kernel
    dim3 blocknums(height);    // Each block processes one row
    dim3 threadsperblock(width); // Each thread processes one column
    softmaxkernel << <blocknums, threadsperblock >> > (dev_input, dev_output, width, height);

    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy output from device to host
    cudaStatus = hipMemcpy(output, dev_output, width * height * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy output failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

Error:
    hipFree(dev_input);
    hipFree(dev_output);
    return cudaStatus;
}
