#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include ""

#define M_PI 3.1415926

__device__ float gelu(float x) {
	return 0.5f * x * (1.0f + tanh(sqrt(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
}

__global__ void gelukernel(float* input, float* output, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		output[idx] = gelu(input[idx]);
	}
}


hipError_t geluWithCuda(float* input, float* output, int N) {
    float* dev_input;
    float* dev_output;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_input, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_input\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_output, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_output\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for input\n");
        return cudaStatus;
    }

    int blockSize = 256; 
    int numBlocks = (N + blockSize - 1) / blockSize;


    gelukernel << <numBlocks, blockSize >> > (dev_input, dev_output, N);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "applyGelu kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(output, dev_output, N * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for output\n");
        return cudaStatus;
    }

    // �ͷ��豸�ڴ�
    hipFree(dev_input);
    hipFree(dev_output);

    return hipSuccess;
}

